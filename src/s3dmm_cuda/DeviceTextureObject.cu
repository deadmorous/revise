#include "s3dmm_cuda/Device3DArray.hpp"
#include "s3dmm_cuda/DeviceArray.hpp"

#include "DeviceTextureObject.hpp"
#include "cudaCheck.hpp"

#include <boost/assert.hpp>

namespace s3dmm {

__host__ DeviceTextureObject::~DeviceTextureObject() {
    free();
}

__host__ void DeviceTextureObject::createBoundTexture(
    const Device3DArray& array,
    bool normalizedCoords,
    hipTextureFilterMode filterMode,
    hipTextureAddressMode addressMode)
{
    free();

    // https://stackoverflow.com/questions/24981310/cuda-create-3d-texture-and-cudaarray3d-from-device-memory
    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array  = array.handle();
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = normalizedCoords;
    texDescr.filterMode = filterMode;
    texDescr.addressMode[0] = addressMode;
    texDescr.addressMode[1] = addressMode;
    texDescr.addressMode[2] = addressMode;
    texDescr.readMode = hipReadModeElementType;
    CU_CHECK(hipCreateTextureObject(&m_t, &texRes, &texDescr, nullptr));
    BOOST_ASSERT(m_t);
}

/*
__host__ void DeviceTextureObject::createBoundTexture(
    const DeviceVector<unsigned char>& v,
    bool normalizedCoords,
    hipTextureFilterMode filterMode,
    hipTextureAddressMode addressMode)
{
    free();

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeLinear;
    texRes.res.linear.devPtr = const_cast<unsigned char*>(v.data());
    texRes.res.linear.desc = hipCreateChannelDesc(8, 8, 8, 8, hipChannelFormatKindUnsigned);
    texRes.res.linear.sizeInBytes = v.size() * sizeof(char);
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = normalizedCoords;
    texDescr.filterMode = filterMode;
    texDescr.addressMode[0] = addressMode;
    texDescr.readMode = hipReadModeElementType;
    CU_CHECK(hipCreateTextureObject(&m_t, &texRes, &texDescr, nullptr));
    BOOST_ASSERT(m_t);
}

__host__ void DeviceTextureObject::createBoundTexture(
    const DeviceVector<float>& v,
    bool normalizedCoords,
    hipTextureFilterMode filterMode,
    hipTextureAddressMode addressMode)
{
    free();

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeLinear;
    texRes.res.linear.devPtr = const_cast<float*>(v.data());
    texRes.res.linear.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    texRes.res.linear.sizeInBytes = v.size() * sizeof(float);
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = normalizedCoords;
    texDescr.filterMode = filterMode;
    texDescr.addressMode[0] = addressMode;
    texDescr.readMode = hipReadModeElementType;
    // texDescr.readMode = hipReadModeNormalizedFloat;
    CU_CHECK(hipCreateTextureObject(&m_t, &texRes, &texDescr, nullptr));
    BOOST_ASSERT(m_t);
}
*/

__host__ void DeviceTextureObject::createBoundTexture(
    const DeviceArray& array,
    bool normalizedCoords,
    hipTextureFilterMode filterMode,
    hipTextureAddressMode addressMode)
{
    free();

    hipResourceDesc    texRes;
    memset(&texRes, 0, sizeof(hipResourceDesc));
    texRes.resType = hipResourceTypeArray;
    texRes.res.array.array = array.handle();
    hipTextureDesc     texDescr;
    memset(&texDescr, 0, sizeof(hipTextureDesc));
    texDescr.normalizedCoords = normalizedCoords;
    texDescr.filterMode = filterMode;
    texDescr.addressMode[0] = addressMode;
    texDescr.readMode = hipReadModeElementType;
    // texDescr.readMode = hipReadModeNormalizedFloat;
    CU_CHECK(hipCreateTextureObject(&m_t, &texRes, &texDescr, nullptr));
    BOOST_ASSERT(m_t);
}

__host__ void DeviceTextureObject::free()
{
    if (m_t) {
        CU_CHECK(hipDestroyTextureObject(m_t));
        m_t = 0;
    }
}

} // s3dmm
