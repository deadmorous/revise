#include "s3dmm_cuda/CudaChannelFormat.hpp"

#include <boost/static_assert.hpp>

struct hipChannelFormatDesc;

namespace s3dmm {

hipChannelFormatDesc CudaChannelFormat::makeCudaChannelFormatDesc() const
{
    return makeCudaChannelFormatDesc(*this);
}

hipChannelFormatDesc CudaChannelFormat::makeCudaChannelFormatDesc(
    const CudaChannelFormat& channelFormat)
{
    BOOST_STATIC_ASSERT(hipChannelFormatKindSigned == Signed);
    BOOST_STATIC_ASSERT(hipChannelFormatKindUnsigned == Unsigned);
    BOOST_STATIC_ASSERT(hipChannelFormatKindFloat == Float);
    BOOST_STATIC_ASSERT(hipChannelFormatKindNone == None);
    return hipCreateChannelDesc(
        channelFormat.x,
        channelFormat.y,
        channelFormat.z,
        channelFormat.w,
        static_cast<hipChannelFormatKind>(channelFormat.f));
}

} // s3dmm
