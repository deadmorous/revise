#include "s3dmm_cuda/manage_profiling.hpp"

#include <hip/hip_runtime_api.h>

#include "cudaCheck.hpp"

namespace s3dmm {

void enableCudaProfiling() {
    CU_CHECK(hipProfilerStart());
}

void disableCudaProfiling() {
    CU_CHECK(hipProfilerStop());
}

}
